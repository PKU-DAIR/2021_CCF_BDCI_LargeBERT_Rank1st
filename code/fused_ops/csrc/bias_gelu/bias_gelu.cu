#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <c10/cuda/CUDAMathCompat.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_fp16.h>
// #include <hip/hip_bf16.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>
#include "util.h"

template <typename acc_t>
__device__ acc_t torch_gelu(acc_t y) {
    return normcdff(y) * y;
}

template <typename acc_t>
__device__ acc_t fast_gelu(acc_t y) {
    return y * 0.5 * (1.0 + tanhf(0.79788456 * y * (1 + 0.044715 * y * y)));
}

template <typename acc_t>
__device__ acc_t torch_gelu_back(acc_t y, acc_t g) {
    constexpr acc_t kBeta = M_2_SQRTPI * M_SQRT1_2 * 0.5;
    const acc_t cdf = normcdff(y);
    const acc_t pdf = expf(-0.5f * y * y) * kBeta;
    return g * (cdf + y * pdf);
}

template <typename acc_t>
__device__ acc_t fast_gelu_back(acc_t y, acc_t g) {
    const acc_t tanh_out = tanhf(0.79788456 * y * (1 + 0.044715 * y * y));
    const acc_t ff = 0.5 * y * ((1 - tanh_out * tanh_out) * (0.79788456 + 0.1070322243 * y * y)) + 0.5 * (1 + tanh_out);
    return ff * g;
}

template <typename index_t, typename input_t, typename output_t, typename acc_t, acc_t (*gelu_func)(acc_t)>
__global__ void bias_gelu_forward(output_t *dst, const input_t *src, const input_t *bias, index_t bsz, int dim) {
    for (int j = threadIdx.x; j < dim; j += blockDim.x) {
        if (blockIdx.x < bsz) {
            const index_t idx = blockIdx.x * dim + j;
            const acc_t y = (acc_t)(src[idx] + bias[j]);
            dst[idx] = (output_t)gelu_func(y);
        }
    }
}

template <typename index_t, typename input_t, typename output_t, typename acc_t, acc_t (*gelu_func)(acc_t)>
__global__ void bias_gelu_forward_vec(output_t *dst, const input_t *src, const input_t *bias, index_t bsz, int dim) {
    using VecInType = VecType<input_t, 2>;
    using VecOutType = VecType<output_t, 2>;
    for (int j = threadIdx.x * 2; j < dim; j += blockDim.x * 2) {
        if (blockIdx.x < bsz) {
            const index_t idx = blockIdx.x * dim + j;
            const VecInType s = *(VecInType *)(src + idx);
            const VecInType b = *(VecInType *)(bias + j);
            const acc_t y1 = s.x + b.x;
            const acc_t y2 = s.y + b.y;
            VecOutType d;
            d.x = gelu_func(y1);
            d.y = gelu_func(y2);
            *(VecOutType *)(dst + idx) = d;
        }
    }
}

template <typename index_t, typename input_t, typename output_t, typename acc_t, acc_t (*gelu_back_func)(acc_t, acc_t)>
__global__ void bias_gelu_backward(output_t *dst, const input_t *src, const input_t *bias,
    const input_t *grad, index_t bsz, int dim) {
    for (int j = threadIdx.x; j < dim; j += blockDim.x) {
        if (blockIdx.x < bsz) {
            const index_t idx = blockIdx.x * dim + j;
            const acc_t y = (acc_t)(src[idx] + bias[j]);
            const acc_t g = grad[idx];
            dst[idx] = (output_t)gelu_back_func(y, g);
        }
    }
}

template <typename index_t, typename input_t, typename output_t, typename acc_t, acc_t (*gelu_back_func)(acc_t, acc_t)>
__global__ void bias_gelu_backward_vec(output_t *dst, const input_t *src, const input_t *bias,
    const input_t *grad, index_t bsz, int dim) {
    using VecInType = VecType<input_t, 2>;
    using VecOutType = VecType<output_t, 2>;
    for (int j = threadIdx.x * 2; j < dim; j += blockDim.x * 2) {
        if (blockIdx.x < bsz) {
            const index_t idx = blockIdx.x * dim + j;
            const VecInType s = *(VecInType *)(src + idx);
            const VecInType b = *(VecInType *)(bias + j);
            const VecInType g = *(VecInType *)(grad + idx);
            const acc_t y1 = s.x + b.x;
            const acc_t y2 = s.y + b.y;
            VecOutType d;
            d.x = gelu_back_func(y1, g.x);
            d.y = gelu_back_func(y2, g.y);
            *(VecOutType *)(dst + idx) = d;
        }
    }
}

template <float (*gelu_func)(float)>
torch::Tensor bias_gelu_forward_cuda(const torch::Tensor &x, const torch::Tensor &bias) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    auto sizes = x.sizes();
    size_t bsz = 1;
    for (size_t i = 0; i + 1 < sizes.size(); ++i) {
        bsz *= sizes[i];
    }
    int dim = sizes[sizes.size() - 1];
    auto dst_options = x.options().requires_grad(false);
    torch::Tensor results = torch::empty(sizes, dst_options);
    auto type = x.scalar_type();
    const int ThreadsPerBlock = 256;
    int ThreadsPerBlockVec = DIV_CELL(dim, 256) * 256 % 512 == 0 ? 256 : 128;
    // if (type == at::ScalarType::BFloat16) {
    //     if (dim % 2 == 0) {
    //         bias_gelu_forward_vec<size_t, hip_bfloat16, hip_bfloat16, float, gelu_func><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
    //             (hip_bfloat16 *)results.data_ptr(),
    //             (const hip_bfloat16 *)x.data_ptr(),
    //             (const hip_bfloat16 *)bias.data_ptr(),
    //             bsz,
    //             dim);
    //     } else {
    //         bias_gelu_forward<size_t, hip_bfloat16, hip_bfloat16, float, gelu_func><<<bsz, ThreadsPerBlock, 0, stream>>>(
    //             (hip_bfloat16 *)results.data_ptr(),
    //             (const hip_bfloat16 *)x.data_ptr(),
    //             (const hip_bfloat16 *)bias.data_ptr(),
    //             bsz,
    //             dim);
    //     }
    // } else if (type == at::ScalarType::Half) {
    //     if (dim % 2 == 0) {
    //         bias_gelu_forward_vec<size_t, half, half, float, gelu_func><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
    //             (half *)results.data_ptr(),
    //             (const half *)x.data_ptr(),
    //             (const half *)bias.data_ptr(),
    //             bsz,
    //             dim);
    //     } else {
    //         bias_gelu_forward<size_t, half, half, float, gelu_func><<<bsz, ThreadsPerBlock, 0, stream>>>(
    //             (half *)results.data_ptr(),
    //             (const half *)x.data_ptr(),
    //             (const half *)bias.data_ptr(),
    //             bsz,
    //             dim);
    //     }
    // } else if (type == at::ScalarType::Float) {
        if (type == at::ScalarType::Float) {
        bias_gelu_forward<size_t, float, float, float, gelu_func><<<bsz, ThreadsPerBlock, 0, stream>>>(
            (float *)results.data_ptr(),
            (const float *)x.data_ptr(),
            (const float *)bias.data_ptr(),
            bsz,
            dim);
    }
    return results;
}

template <float (*gelu_back_func)(float, float)>
torch::Tensor bias_gelu_backward_cuda(const torch::Tensor &x, const torch::Tensor &bias, const torch::Tensor &grad) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    auto sizes = x.sizes();
    size_t bsz = 1;
    for (size_t i = 0; i + 1 < sizes.size(); ++i) {
        bsz *= sizes[i];
    }
    int dim = sizes[sizes.size() - 1];
    auto dst_options = x.options().requires_grad(false);
    torch::Tensor results = torch::empty(sizes, dst_options);
    auto type = x.scalar_type();
    const int ThreadsPerBlock = 256;
    int ThreadsPerBlockVec = DIV_CELL(dim, 256) * 256 % 512 == 0 ? 256 : 128;
    // if (type == at::ScalarType::BFloat16) {
    //     if (dim % 2 == 0) {
    //         bias_gelu_backward_vec<size_t, hip_bfloat16, hip_bfloat16, float, gelu_back_func><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
    //             (hip_bfloat16 *)results.data_ptr(),
    //             (const hip_bfloat16 *)x.data_ptr(),
    //             (const hip_bfloat16 *)bias.data_ptr(),
    //             (const hip_bfloat16 *)grad.data_ptr(),
    //             bsz,
    //             dim);
    //     } else {
    //         bias_gelu_backward<size_t, hip_bfloat16, hip_bfloat16, float, gelu_back_func><<<bsz, ThreadsPerBlock, 0, stream>>>(
    //             (hip_bfloat16 *)results.data_ptr(),
    //             (const hip_bfloat16 *)x.data_ptr(),
    //             (const hip_bfloat16 *)bias.data_ptr(),
    //             (const hip_bfloat16 *)grad.data_ptr(),
    //             bsz,
    //             dim);
    //         }
    // } else if (type == at::ScalarType::Half) {
    //     if (dim % 2 == 0) {
    //         bias_gelu_backward_vec<size_t, half, half, float, gelu_back_func><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
    //             (half *)results.data_ptr(),
    //             (const half *)x.data_ptr(),
    //             (const half *)bias.data_ptr(),
    //             (const half *)grad.data_ptr(),
    //             bsz,
    //             dim);
    //     } else {
    //         bias_gelu_backward<size_t, half, half, float, gelu_back_func><<<bsz, ThreadsPerBlock, 0, stream>>>(
    //             (half *)results.data_ptr(),
    //             (const half *)x.data_ptr(),
    //             (const half *)bias.data_ptr(),
    //             (const half *)grad.data_ptr(),
    //             bsz,
    //             dim);
    //     }
    // } else if (type == at::ScalarType::Float) {
        if (type == at::ScalarType::Float) {
        bias_gelu_backward<size_t, float, float, float, gelu_back_func><<<bsz, ThreadsPerBlock, 0, stream>>>(
            (float *)results.data_ptr(),
            (const float *)x.data_ptr(),
            (const float *)bias.data_ptr(),
            (const float *)grad.data_ptr(),
            bsz,
            dim);
    }
    return results;
}

using ForwardFunc = torch::Tensor (*)(const torch::Tensor &, const torch::Tensor &);

ForwardFunc bias_gelu_torch_forward_cuda = bias_gelu_forward_cuda<torch_gelu>;
ForwardFunc bias_gelu_fast_forward_cuda = bias_gelu_forward_cuda<fast_gelu>;

using BackwardFunc = torch::Tensor (*)(const torch::Tensor &, const torch::Tensor &, const torch::Tensor &);

BackwardFunc bias_gelu_torch_backward_cuda = bias_gelu_backward_cuda<torch_gelu_back>;
BackwardFunc bias_gelu_fast_backward_cuda = bias_gelu_backward_cuda<fast_gelu_back>;