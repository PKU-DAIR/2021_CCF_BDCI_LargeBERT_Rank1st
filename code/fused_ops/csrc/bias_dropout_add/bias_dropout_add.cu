#include "hip/hip_runtime.h"
#include <vector>
#include <ATen/ATen.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <c10/cuda/CUDAMathCompat.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_fp16.h>
// #include <hip/hip_bf16.h>
#include <hiprand/hiprand_kernel.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>
#include "util.h"

template <typename MaskType, typename acc_t, typename IndexType>
__global__ void generate_dropout_mask_kernel(MaskType* output, IndexType n, uint64_t seed, uint64_t offset, acc_t p) {
    IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed, idx * sizeof(MaskType) * 8, offset, &state);
    MaskType mask = 0;
    #pragma unroll
    for (int i = 0; i < sizeof(MaskType) * 2; ++i) {
        const float4 rand = hiprand_uniform4(&state);
        mask |= (((MaskType)(rand.x < p)) << (i * 4))
            | (((MaskType)(rand.y < p)) << (i * 4 + 1))
            | (((MaskType)(rand.z < p)) << (i * 4 + 2))
            | (((MaskType)(rand.w < p)) << (i * 4 + 3));
    }
    if (idx < n) {
        output[idx] = mask;
    }
}

template <typename MaskType>
void generate_dropout_mask(MaskType* mask, int bsz, int dim, float p, uint64_t seed, uint64_t offset) {
    const int mask_elements_per_batch = DIV_CELL(dim, sizeof(MaskType) * 8);
    const int num_elements = bsz * mask_elements_per_batch;
    const int block_size = 128;
    const int grid = DIV_CELL(num_elements, block_size);
    generate_dropout_mask_kernel<MaskType, float, size_t><<<grid, block_size>>>(mask, num_elements, seed, offset, p);
}

template <typename T>
__device__ __forceinline__ T from_uint8(uint8_t input) {
    return (T)input;
}

// template <>
// __device__ __forceinline__ __hip_bfloat16 from_uint8(uint8_t input) {
//     return (__hip_bfloat16)(float)input;
// }

template <typename index_t, typename input_t, typename output_t, bool is_training>
__global__ void bias_dropout_add_forward(output_t *dst, const input_t *x, const input_t *bias,
    const input_t *residual, const uint8_t *mask, index_t bsz, int dim, input_t pinv) {
    if (blockIdx.x < bsz) {
        if IF_CONSTEXPR (is_training) {
            const int mask_index = blockIdx.x * DIV_CELL(dim, 8);
            const uint8_t mask_offset = threadIdx.x % 8;
            for (int j = threadIdx.x; j < dim; j += blockDim.x) {
                const index_t idx = blockIdx.x * dim + j;
                const input_t y = x[idx] + bias[j];
                const input_t m = from_uint8<input_t>((mask[mask_index + j / 8] >> mask_offset) & 1);
                dst[idx] = y * m * pinv + residual[idx];
            }
        } else {
            for (int j = threadIdx.x; j < dim; j += blockDim.x) {
                const index_t idx = blockIdx.x * dim + j;
                dst[idx] = x[idx] + bias[j] + residual[idx];
            }
        }
    }
}

// template <typename index_t, typename input_t, typename output_t, bool is_training>
// __global__ void bias_dropout_add_forward_vec(output_t *dst, const input_t *x, const input_t *bias,
//     const input_t *residual, const uint8_t *mask, index_t bsz, int dim, input_t pinv) {
//     using VecInType = VecType<input_t, 2>;
//     using VecOutType = VecType<output_t, 2>;
//     if (blockIdx.x < bsz) {
//         if IF_CONSTEXPR (is_training) {
//             const int mask_index = blockIdx.x * DIV_CELL(dim, 8);
//             const uint8_t mask_offset1 = (threadIdx.x * 2) % 8;
//             const uint8_t mask_offset2 = (threadIdx.x * 2 + 1) % 8;
//             for (int j = threadIdx.x * 2; j < dim; j += blockDim.x * 2) {
//                 const index_t idx = blockIdx.x * dim + j;
//                 const VecInType xi = *(VecInType *)(x + idx);
//                 const VecInType b = *(VecInType *)(bias + j);
//                 const VecInType r = *(VecInType *)(residual + idx);
//                 const uint8_t m = mask[mask_index + j / 8];
//                 const input_t m1 = from_uint8<input_t>((m >> mask_offset1) & 1);
//                 const input_t m2 = from_uint8<input_t>((m >> mask_offset2) & 1);
//                 VecOutType d;
//                 d.x = (xi.x + b.x) * m1 * pinv + r.x;
//                 d.y = (xi.y + b.y) * m2 * pinv + r.y;
//                 *(VecOutType *)(dst + idx) = d;
//             }
//         } else {
//             for (int j = threadIdx.x * 2; j < dim; j += blockDim.x * 2) {
//                 const index_t idx = blockIdx.x * dim + j;
//                 const VecInType xi = *(VecInType *)(x + idx);
//                 const VecInType b = *(VecInType *)(bias + j);
//                 const VecInType r = *(VecInType *)(residual + idx);
//                 VecOutType d;
//                 d.x = xi.x + b.x + r.x;
//                 d.y = xi.y + b.y + r.y;
//                 *(VecOutType *)(dst + idx) = d;
//             }
//         }
//     }
// }

template <typename index_t, typename input_t, typename output_t>
__global__ void bias_dropout_add_backward(output_t *dst, const input_t *grad, const uint8_t *mask, index_t bsz, int dim, input_t pinv) {
    if (blockIdx.x < bsz) {
        const int mask_index = blockIdx.x * DIV_CELL(dim, 8);
        const uint8_t mask_offset = threadIdx.x % 8;
        for (int j = threadIdx.x; j < dim; j += blockDim.x) {
            const index_t idx = blockIdx.x * dim + j;
            uint8_t m = (mask[mask_index + j / 8] >> mask_offset) & 1;
            dst[idx] = grad[idx] * from_uint8<input_t>(m) * pinv;
        }
    }
}

template <typename index_t, typename input_t, typename output_t>
__global__ void bias_dropout_add_backward_vec(output_t *dst, const input_t *grad, const uint8_t *mask, index_t bsz, int dim, input_t pinv) {
    using VecInType = VecType<input_t, 2>;
    using VecOutType = VecType<output_t, 2>;
    if (blockIdx.x < bsz) {
        const int mask_index = blockIdx.x * DIV_CELL(dim, 8);
        const uint8_t mask_offset1 = (threadIdx.x * 2) % 8;
        const uint8_t mask_offset2 = (threadIdx.x * 2 + 1) % 8;
        for (int j = threadIdx.x * 2; j < dim; j += blockDim.x * 2) {
            const index_t idx = blockIdx.x * dim + j;
            const uint8_t m = mask[mask_index + j / 8];
            const VecInType g = *(VecInType *)(grad + idx);
            VecOutType d;
            d.x = g.x * from_uint8<input_t>((m >> mask_offset1) & 1) * pinv;
            d.y = g.y * from_uint8<input_t>((m >> mask_offset2) & 1) * pinv;
            *(VecOutType *)(dst + idx) = d;
        }
    }
}

std::vector<c10::optional<torch::Tensor>> bias_dropout_add_forward_cuda(const torch::Tensor &x, const torch::Tensor &bias,
    const torch::Tensor &residual, bool is_training, float dropout_prob, c10::optional<at::Generator> gen_) {
    using MaskType = uint64_t;
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    auto sizes = x.sizes();
    size_t bsz = 1;
    for (size_t i = 0; i + 1 < sizes.size(); ++i) {
        bsz *= sizes[i];
    }
    const int dim = sizes[sizes.size() - 1];
    auto dst_options = x.options().requires_grad(false);
    torch::Tensor results = torch::empty(sizes, dst_options);
    auto type = x.scalar_type();
    const int ThreadsPerBlock = 256;
    int ThreadsPerBlockVec = DIV_CELL(dim, 256) * 256 % 512 == 0 ? 256 : 128;
    if (is_training && dropout_prob != 0.0) {
        auto mask_options = dst_options.dtype(torch::kInt64);
        torch::Tensor mask = torch::empty(bsz * DIV_CELL(dim, sizeof(MaskType) * 8), mask_options);
        auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(gen_, at::cuda::detail::getDefaultCUDAGenerator());
        std::pair<uint64_t, uint64_t> rng_engine_inputs;
        {
            // See Note [Acquire lock when using random generators]
            std::lock_guard<std::mutex> lock(gen->mutex_);
            rng_engine_inputs = gen->philox_engine_inputs(sizeof(MaskType) * 8);
        }
        uint64_t seed = std::get<0>(rng_engine_inputs);
        uint64_t offset = std::get<1>(rng_engine_inputs);
        generate_dropout_mask<MaskType>((MaskType *)mask.data_ptr(), bsz, dim, 1.0 - dropout_prob, seed, offset);
        // if (type == at::ScalarType::BFloat16) {
        //     if (dim % 2 == 0) {
        //         bias_dropout_add_forward_vec<size_t, hip_bfloat16, hip_bfloat16, true><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
        //             (hip_bfloat16 *)results.data_ptr(),
        //             (const hip_bfloat16 *)x.data_ptr(),
        //             (const hip_bfloat16 *)bias.data_ptr(),
        //             (const hip_bfloat16 *)residual.data_ptr(),
        //             (const uint8_t *)mask.data_ptr(),
        //             bsz,
        //             dim,
        //             1.0 / (1.0 - dropout_prob));
        //     } else {
        //         bias_dropout_add_forward<size_t, hip_bfloat16, hip_bfloat16, true><<<bsz, ThreadsPerBlock, 0, stream>>>(
        //             (hip_bfloat16 *)results.data_ptr(),
        //             (const hip_bfloat16 *)x.data_ptr(),
        //             (const hip_bfloat16 *)bias.data_ptr(),
        //             (const hip_bfloat16 *)residual.data_ptr(),
        //             (const uint8_t *)mask.data_ptr(),
        //             bsz,
        //             dim,
        //             1.0 / (1.0 - dropout_prob));
        //     }
        // } else if (type == at::ScalarType::Half) {
        //     if (dim % 2 == 0) {
        //         bias_dropout_add_forward_vec<size_t, half, half, true><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
        //             (half *)results.data_ptr(),
        //             (const half *)x.data_ptr(),
        //             (const half *)bias.data_ptr(),
        //             (const half *)residual.data_ptr(),
        //             (const uint8_t *)mask.data_ptr(),
        //             bsz,
        //             dim,
        //             1.0 / (1.0 - dropout_prob));
        //     } else {
        //         bias_dropout_add_forward<size_t, half, half, true><<<bsz, ThreadsPerBlock, 0, stream>>>(
        //             (half *)results.data_ptr(),
        //             (const half *)x.data_ptr(),
        //             (const half *)bias.data_ptr(),
        //             (const half *)residual.data_ptr(),
        //             (const uint8_t *)mask.data_ptr(),
        //             bsz,
        //             dim,
        //             1.0 / (1.0 - dropout_prob));
        //     }
        // } else if (type == at::ScalarType::Float) {
            if (type == at::ScalarType::Float) {
            bias_dropout_add_forward<size_t, float, float, true><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (float *)results.data_ptr(),
                (const float *)x.data_ptr(),
                (const float *)bias.data_ptr(),
                (const float *)residual.data_ptr(),
                (const uint8_t *)mask.data_ptr(),
                bsz,
                dim,
                1.0 / (1.0 - dropout_prob));
        }
        return {results, mask};
    } else {
        // if (type == at::ScalarType::BFloat16) {
        //     if (dim % 2 == 0) {
        //         bias_dropout_add_forward_vec<size_t, hip_bfloat16, hip_bfloat16, false><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
        //             (hip_bfloat16 *)results.data_ptr(),
        //             (const hip_bfloat16 *)x.data_ptr(),
        //             (const hip_bfloat16 *)bias.data_ptr(),
        //             (const hip_bfloat16 *)residual.data_ptr(),
        //             nullptr,
        //             bsz,
        //             dim,
        //             0.0);
        //     } else {
        //         bias_dropout_add_forward<size_t, hip_bfloat16, hip_bfloat16, false><<<bsz, ThreadsPerBlock, 0, stream>>>(
        //             (hip_bfloat16 *)results.data_ptr(),
        //             (const hip_bfloat16 *)x.data_ptr(),
        //             (const hip_bfloat16 *)bias.data_ptr(),
        //             (const hip_bfloat16 *)residual.data_ptr(),
        //             nullptr,
        //             bsz,
        //             dim,
        //             0.0);
        //     }
        // } else if (type == at::ScalarType::Half) {
        //     if (dim % 2 == 0) {
        //         bias_dropout_add_forward_vec<size_t, half, half, false><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
        //             (half *)results.data_ptr(),
        //             (const half *)x.data_ptr(),
        //             (const half *)bias.data_ptr(),
        //             (const half *)residual.data_ptr(),
        //             nullptr,
        //             bsz,
        //             dim,
        //             0.0);
        //     } else {
        //         bias_dropout_add_forward<size_t, half, half, false><<<bsz, ThreadsPerBlock, 0, stream>>>(
        //             (half *)results.data_ptr(),
        //             (const half *)x.data_ptr(),
        //             (const half *)bias.data_ptr(),
        //             (const half *)residual.data_ptr(),
        //             nullptr,
        //             bsz,
        //             dim,
        //             0.0);
        //     }
        // } else if (type == at::ScalarType::Float) {
            if (type == at::ScalarType::Float) {
            bias_dropout_add_forward<size_t, float, float, false><<<bsz, ThreadsPerBlock, 0, stream>>>(
                (float *)results.data_ptr(),
                (const float *)x.data_ptr(),
                (const float *)bias.data_ptr(),
                (const float *)residual.data_ptr(),
                nullptr,
                bsz,
                dim,
                0.0);
        }
        return {results, c10::optional<torch::Tensor>()};
    }
}

torch::Tensor bias_dropout_add_backward_cuda(const torch::Tensor &grad, const torch::Tensor &mask, float dropout_prob) {
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    auto sizes = grad.sizes();
    size_t bsz = 1;
    for (size_t i = 0; i + 1 < sizes.size(); ++i) {
        bsz *= sizes[i];
    }
    int dim = sizes[sizes.size() - 1];
    auto dst_options = grad.options().requires_grad(false);
    torch::Tensor results = torch::empty(sizes, dst_options);
    auto type = grad.scalar_type();
    const int ThreadsPerBlock = 256;
    int ThreadsPerBlockVec = DIV_CELL(dim, 256) * 256 % 512 == 0 ? 256 : 128;
    // if (type == at::ScalarType::BFloat16) {
    //     if (dim % 2 == 0) {
    //         bias_dropout_add_backward_vec<size_t, hip_bfloat16, hip_bfloat16><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
    //             (hip_bfloat16 *)results.data_ptr(),
    //             (const hip_bfloat16 *)grad.data_ptr(),
    //             (const uint8_t *)mask.data_ptr(),
    //             bsz,
    //             dim,
    //             1.0 / (1.0 - dropout_prob));
    //     } else {
    //         bias_dropout_add_backward<size_t, hip_bfloat16, hip_bfloat16><<<bsz, ThreadsPerBlock, 0, stream>>>(
    //             (hip_bfloat16 *)results.data_ptr(),
    //             (const hip_bfloat16 *)grad.data_ptr(),
    //             (const uint8_t *)mask.data_ptr(),
    //             bsz,
    //             dim,
    //             1.0 / (1.0 - dropout_prob));
    //     }
    // } else if (type == at::ScalarType::Half) {
    //     if (dim % 2 == 0) {
    //         bias_dropout_add_backward_vec<size_t, half, half><<<bsz, ThreadsPerBlockVec, 0, stream>>>(
    //             (half *)results.data_ptr(),
    //             (const half *)grad.data_ptr(),
    //             (const uint8_t *)mask.data_ptr(),
    //             bsz,
    //             dim,
    //             1.0 / (1.0 - dropout_prob));
    //     } else {
    //         bias_dropout_add_backward<size_t, half, half><<<bsz, ThreadsPerBlock, 0, stream>>>(
    //             (half *)results.data_ptr(),
    //             (const half *)grad.data_ptr(),
    //             (const uint8_t *)mask.data_ptr(),
    //             bsz,
    //             dim,
    //             1.0 / (1.0 - dropout_prob));
    //     }
    // } else if (type == at::ScalarType::Float) {
        if (type == at::ScalarType::Float) {
        bias_dropout_add_backward<size_t, float, float><<<bsz, ThreadsPerBlock, 0, stream>>>(
            (float *)results.data_ptr(),
            (const float *)grad.data_ptr(),
            (const uint8_t *)mask.data_ptr(),
            bsz,
            dim,
            1.0 / (1.0 - dropout_prob));
    }
    return results;
}